
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(int *a, int *b, int *c, int n){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n)
		for(int j=0;j<100;j++)
			c[i] = c[i] + a[i]*b[i];
}

int main(void){
	int * a, * b;
	int * d_a, * d_b;
	int * d_r1, * d_r2, *d_r3;
	int * temp, * temp2;
	const int n = 1<<24;
	const int n_s = 3;

	hipStream_t streams[n_s];
	for(int i=0;i<n_s;i++)
		hipStreamCreate(&streams[i]);

	a = new int[n*sizeof(int)];
	b = new int[n*sizeof(int)];
	temp = new int[n*sizeof(int)];
	temp2 = new int[n*sizeof(int)];

	hipMalloc(&d_a, n*sizeof(int));
       	hipMalloc(&d_b, n*sizeof(int));
	hipMalloc(&d_r1, n*sizeof(int));
	hipMalloc(&d_r2, n*sizeof(int));
	hipMalloc(&d_r3, n*sizeof(int));

	for(int i=0;i<n;i++){
		a[i] = 3;
		b[i] = 5;
	}

	int blockSize = 256;
	int numBlocks = n/256;
	
	hipMemcpyAsync(d_a, a, n*sizeof(int), hipMemcpyHostToDevice, streams[0]);
	vectorAdd<<<numBlocks,blockSize,0,streams[0]>>>(d_a,d_a,d_r1,n);

	hipMemcpyAsync(d_b, b, n*sizeof(int), hipMemcpyHostToDevice, streams[1]);
	vectorAdd<<<numBlocks, blockSize,0,streams[1]>>>(d_b,d_b,d_r2,n);

	vectorAdd<<<numBlocks, blockSize,0,streams[2]>>>(d_a,d_b,d_r3,n);
	
	hipMemcpyAsync(temp, d_r1, n*sizeof(int), hipMemcpyDeviceToHost, streams[0]);
	hipDeviceSynchronize();
	temp2[0] = temp[0];
	for(int i=1;i<n;i++)
		temp2[i] = temp2[i-1] + temp[i];
	hipMemcpyAsync(temp, d_r2, n*sizeof(int), hipMemcpyDeviceToHost, streams[1]);
	hipDeviceSynchronize();
	temp2[0] = temp[0];
	for(int i=1;i<n;i++)
		temp2[i] = temp2[i-1] + temp[i];
	hipMemcpyAsync(temp, d_r3, n*sizeof(int), hipMemcpyDeviceToHost, streams[2]);	
	hipDeviceSynchronize();
	temp2[0] = temp[0];
	for(int i=1;i<n;i++)
		temp2[i] = temp2[i-1] + temp[i];

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_r1);
	hipFree(d_r2);
	hipFree(d_r3);
	delete a;
	delete b;
	delete [] temp;
	delete [] temp2;

	return 0;
}
