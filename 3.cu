
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(int *a, int *b, int *c, int n){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n)
		for(int j=0;j<100;j++)
			c[i] = a[i] + b[i];
}

int main(void){
	int * a, * b;
	int * r1, * r2, *r3;
	int * temp;
	const int n = 1<<24;
	const int n_s = 3;

	hipStream_t streams[n_s];
	for(int i=0;i<n_s;i++)
		hipStreamCreate(&streams[i]);

	hipMallocManaged(&a, n*sizeof(int));
       	hipMallocManaged(&b, n*sizeof(int));
	hipMallocManaged(&r1, n*sizeof(int));
	hipMallocManaged(&r2, n*sizeof(int));
	hipMallocManaged(&r3, n*sizeof(int));
	temp = new int[n*sizeof(int)];

	for(int i=0;i<n;i++){
		a[i] = 3;
		b[i] = 5;
	}

	int blockSize = 256;
	int numBlocks = n/256;
	
	vectorAdd<<<numBlocks,blockSize,0,streams[0]>>>(a,a,r1,n);
	
	vectorAdd<<<numBlocks, blockSize,0,streams[1]>>>(b,b,r2,n);
	
	vectorAdd<<<numBlocks, blockSize,0,streams[2]>>>(a,b,r3,n);

	hipDeviceSynchronize();
	temp[0] = r1[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1] + r1[i];
	hipDeviceSynchronize();
	temp[0] = r2[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1] + r2[i];
	hipDeviceSynchronize();
	temp[0] = r3[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1] + r3[i];

	hipFree(a);
	hipFree(b);
	hipFree(r1);
	hipFree(r2);
	hipFree(r3);
	delete [] temp;
	
	return 0;
}
