
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(int *a, int *b, int *c, int n){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n)
		for(int j=0;j<100;j++)
			c[i] = a[i] + b[i];
}

int main(void){
	int * a, * b, * c;
	int * temp;
	int n = 1<<24;

	hipMallocManaged(&a, n*sizeof(int));
       	hipMallocManaged(&b, n*sizeof(int));
	hipMallocManaged(&c, n*sizeof(int));
	temp = new int[n*sizeof(int)];

	for(int i=0;i<n;i++){
		a[i] = 3;
		b[i] = 5;
	}

	int blockSize = 256;
	int numBlocks = n/256;

	vectorAdd<<<numBlocks,blockSize>>>(a,b,c,n);
	hipDeviceSynchronize();

	temp[0] = a[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1] + a[i];

	vectorAdd<<<numBlocks,blockSize>>>(a,b,c,n);
	hipDeviceSynchronize();

	temp[0] = a[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1] + a[i];
	temp[0] = b[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1] + b[i];

	vectorAdd<<<numBlocks, blockSize>>>(a,b,c,n);
	hipDeviceSynchronize();

	temp[0] = c[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1] + c[i];

	vectorAdd<<<numBlocks, blockSize>>>(a,b,c,n);
	hipDeviceSynchronize();

	vectorAdd<<<numBlocks, blockSize>>>(a,b,c,n);
	hipDeviceSynchronize();
	
	hipFree(a);
	hipFree(b);
	hipFree(c);
	delete temp;

	return 0;
}
