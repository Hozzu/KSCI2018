
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(int *a, int *b, int *c, int n){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n)
		for(int j=0;j<100;j++)
			c[i] = a[i] + b[i];
}

int main(void){
	int * a, * b, * c;
	int * d_a, * d_b, * d_c;
	int * temp;
	int n = 1<<24;

	a = new int[n*sizeof(int)];
	b = new int[n*sizeof(int)];
	c = new int[n*sizeof(int)];
	temp = new int[n*sizeof(int)];

	hipMalloc(&d_a, n*sizeof(int));
       	hipMalloc(&d_b, n*sizeof(int));
	hipMalloc(&d_c, n*sizeof(int));

	for(int i=0;i<n;i++){
		a[i] = 3;
		b[i] = 5;
	}

	int blockSize = 256;
	int numBlocks = n/256;

	hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice);
	vectorAdd<<<numBlocks,blockSize>>>(d_a,d_b,d_c,n);
	hipDeviceSynchronize();

	temp[0] = a[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1]+a[i];

	vectorAdd<<<numBlocks, blockSize>>>(d_a,d_b,d_c,n);
	hipDeviceSynchronize();

	temp[0] = a[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1]+a[i];
	temp[0] = b[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1]+b[i];

	vectorAdd<<<numBlocks, blockSize>>>(d_a,d_b,d_c,n);
	hipDeviceSynchronize();

	hipMemcpy(c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);	
	temp[0] = c[0];
	for(int i=1;i<n;i++)
		temp[i] = temp[i-1]+c[i];

	vectorAdd<<<numBlocks, blockSize>>>(d_a,d_b,d_c,n);
	hipDeviceSynchronize();

	vectorAdd<<<numBlocks, blockSize>>>(d_a,d_b,d_c,n);
	hipDeviceSynchronize();

	hipFree(a);
	hipFree(b);
	hipFree(c);
	delete temp;

	return 0;
}
